
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
{
#define AUXILIARY_NODE_ID -1

    typedef struct node
    {
        int id;
        node *prev;
        node *next;
        __device__ node(int id) : id(id){};
    } node_t;

    __global__ void init_rng(int nthreads, hiprandState *states, unsigned long long seed, unsigned long long offset)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id >= nthreads)
            return;
        /* Each thread gets same seed, a different sequence number, no offset */
        hiprand_init(seed, id, offset, &states[id]);
    }

    __global__ void generate_rr_sets(float *data, int *rows, int *cols, bool *out, int numNodes, int numNonZeros, int numSets, hiprandState *states)
    {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid < numSets)
        {
            hiprandState state = states[tid];

            /* Because C does not give us the luxury of dynamic arrays, to imitate the
        behavior of a stack, I am using a linked list*/
            int randomNodeId = ceil(numNodes * hiprand_uniform(&state)) - 1;
            node *stack = new node(randomNodeId);
            node *auxiliary = new node(AUXILIARY_NODE_ID);
            auxiliary->next = stack;
            stack->prev = auxiliary;

            // Returns false when stack is NULL
            while (stack->id != AUXILIARY_NODE_ID)
            {
                // pop from stack
                int currentNodeId = stack->id;
                node *temp = stack;
                stack = stack->prev;
                free(temp);

                // If current is not in visited
                if (!out[tid * numNodes + currentNodeId])
                {
                    out[tid * numNodes + currentNodeId] = true;

                    int dataStart = rows[currentNodeId];
                    int dataEnd = rows[currentNodeId + 1];

                    for (unsigned int i = dataStart; i < dataEnd; i++)
                    {
                        if (hiprand_uniform(&state) < data[i])
                        {
                            // append to stack
                            stack->next = new node(cols[i]);
                            stack->next->prev = stack;
                            stack = stack->next;
                        }
                    }
                }
            }
            free(auxiliary);
        }
    }
}